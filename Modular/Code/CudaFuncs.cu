#include "hip/hip_runtime.h"
//i will write the cuda functions here
#include "../Project.hpp"
//this should set to[i] to the sum of from[i]
//the first integer in from[i] (aka from[i][0])
//is the size of that array.
extern "C" __global__ void vectorSum(int *to, int **from, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        int *current_from_array = from[idx];
        
        // Ensure the pointer is not null before dereferencing
        if (current_from_array != nullptr) {
            int num_elements_to_sum = current_from_array[0];
            int current_sum = 0;
            
            // Sum elements from current_from_array[1] to current_from_array[num_elements_to_sum]
            for (int j = 1; j <= num_elements_to_sum; j++) {
                current_sum += current_from_array[j];
            }
            to[idx] = current_sum;
        }
    }
}

namespace Wrapper{

void VSWrapper(int *to, int **from, int size) {
    // Define the number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vectorSum<<<blocksPerGrid, threadsPerBlock>>>(to, from, size);
    // Check for errors in kernel launch (optional)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
	err = hipDeviceSynchronize(); // Wait for kernel to complete
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
}
}